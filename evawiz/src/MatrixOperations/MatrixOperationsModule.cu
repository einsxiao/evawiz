#include "hip/hip_runtime.h"
#include<evawiz.h>
#include"MatrixOperationsModule.h"
#include"MPIModule.h"
#include"hipfft/hipfft.h"

#include<fftw3-mpi.h>

using namespace std;
using namespace evawiz;
#define ModuleName MatrixOperations
__DEBUG_MAIN__("input.ev");

MPIModule* evawiz::MatrixOperationsModule::mpiModule = NULL;
ModuleInitialize
{
  mpiModule = (evawiz::MPIModule*)(EvaKernel->ModulePtr("MPI"));

}

ModuleFinalize
{

} 

void MatrixOperationsModule::Init()
{

}

const int PlanType_gpu_cpu = 0;
const int PlanType_mpi = 1;
const int PlanType_float_gpu_cpu = 2;
const int PlanType_float_mpi = 3;

static bool mpiFFTInited = false;
static void mpi_fft_init()
{
  if ( not mpiFFTInited ){
    mpiFFTInited = true;
    fftw_mpi_init();
  }
}

template<class type>
void MPI_FFT_MatrixInit(Matrix_T<type>&mat,int rank, int n[],int ttype,int batch)
{
  mpi_fft_init();
  ptrdiff_t alloc_local, local_n0, local_0_start,nn[3];
  long dist = 1;
  for (int i=0;i<rank; i++){
    dist *= n[i];
    nn[i] = n[i];
  }
  if ( ttype == PlanType_mpi ){
    alloc_local = fftw_mpi_local_size_many( rank, nn, batch, dist, MPI_COMM_WORLD , &local_n0, &local_0_start);
  }else{
    alloc_local = fftwf_mpi_local_size_many( rank, nn, batch, dist, MPI_COMM_WORLD , &local_n0, &local_0_start);
  }
  mat.Init(1,alloc_local,MatrixHostDevice);
  //mat.ReShape(3, n1/mpiModule->rankSize, n2, n3);
}

class planRecord
{
public:
  int                 rank;
  int                 type; //
  int                 batch;
  int                 n[3];
  hipfftHandle         gpu_plan;
  fftw_plan           cpu_plan_forward, cpu_plan_backward;
  fftwf_plan          fcpu_plan_forward, fcpu_plan_backward;
  ComplexMatrix       mat;
  FloatComplexMatrix  fmat;
  planRecord(int rank, int *inn, int type, int batch=1)
  {
    this->rank = rank;
    this->type = type;
    this->batch = batch;
    for ( int i=0;i<rank;i++)
      this->n[i] = inn[i];
    int stride = 1, dist = n[0]*n[1]*n[2];
    //if mpi plan 
    if ( type == PlanType_mpi ){ //mpi
      MPI_FFT_MatrixInit(mat, rank, n,type, batch );
      ptrdiff_t nn[3]; for ( int i=0;i<rank;i++) nn[i] = n[i];
      cpu_plan_forward = fftw_mpi_plan_many_dft(rank, nn, batch, dist, dist, (fftw_complex*)mat.Data, (fftw_complex*)mat.Data, MPI_COMM_WORLD, FFTW_FORWARD, FFTW_ESTIMATE );
      cpu_plan_backward = fftw_mpi_plan_many_dft(rank, nn, batch, dist, dist, (fftw_complex*)mat.Data, (fftw_complex*)mat.Data, MPI_COMM_WORLD, FFTW_BACKWARD, FFTW_ESTIMATE );
    }else if ( type == PlanType_float_mpi ){ //float mpi
      MPI_FFT_MatrixInit(fmat, rank, n, type, batch );
      ptrdiff_t nn[3]; for ( int i=0;i<rank;i++) nn[i] = n[i];
      fcpu_plan_forward = fftwf_mpi_plan_many_dft(rank, nn, batch, dist, dist, (fftwf_complex*)fmat.Data, (fftwf_complex*)fmat.Data, MPI_COMM_WORLD, FFTW_FORWARD, FFTW_ESTIMATE );
      fcpu_plan_backward = fftwf_mpi_plan_many_dft(rank, nn, batch, dist, dist, (fftwf_complex*)fmat.Data, (fftwf_complex*)fmat.Data, MPI_COMM_WORLD, FFTW_BACKWARD, FFTW_ESTIMATE );
    }else if ( type == PlanType_gpu_cpu ){// gpu cpu
      mat.Init(1,batch*dist,MatrixHostDevice);
      if ( EvaSettings::RunningMode() == RunningModeGpu ){
        hipfftPlanMany(&gpu_plan,rank,n,n,stride,dist,n,stride,dist,HIPFFT_Z2Z,batch);
        CUDA_LAST_ERROR();
      }
      cpu_plan_forward = fftw_plan_many_dft( rank, inn, batch, (fftw_complex*)mat.Data, n, stride, dist, (fftw_complex*)mat.Data, n, stride, dist, FFTW_FORWARD, FFTW_ESTIMATE ); 
      cpu_plan_backward = fftw_plan_many_dft( rank, inn, batch, (fftw_complex*)mat.Data, n, stride, dist, (fftw_complex*)mat.Data, n, stride, dist, FFTW_BACKWARD, FFTW_ESTIMATE ); 
    }else{// float gpu cpu
      fmat.Init(1,batch*dist,MatrixHostDevice);
      if ( EvaSettings::RunningMode() ==  RunningModeGpu ){
        hipfftPlanMany(&gpu_plan,rank,n,n,stride,dist,n,stride,dist,HIPFFT_C2C,batch);
        CUDA_LAST_ERROR();
      }
      fcpu_plan_forward = fftwf_plan_many_dft( rank, inn, batch, (fftwf_complex*)fmat.Data, n, stride, dist, (fftwf_complex*)fmat.Data, n, stride, dist, FFTW_FORWARD, FFTW_ESTIMATE ); 
      fcpu_plan_backward = fftwf_plan_many_dft( rank, inn, batch, (fftwf_complex*)fmat.Data, n, stride, dist, (fftwf_complex*)fmat.Data, n, stride, dist, FFTW_BACKWARD, FFTW_ESTIMATE ); 
    }

  }
  ~planRecord()
  {
  }
  bool planQ(int rank, int inn[], int type, int batch=1)
  {
    if ( n[0] == inn[0] and n[1] == inn[1] and n[2] == inn[2] and this->batch == batch and this->type == type )
      return true;
    return false;
  }
  void execute(ComplexMatrix &in_mat,ComplexMatrix &out_mat,MatrixOperationsType dire)
  {
    if ( type == PlanType_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      if ( dire == MatrixOperations_FFT_FORWARD )
        hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*) in_mat.DataDevice, (hipfftDoubleComplex*) out_mat.DataDevice, HIPFFT_FORWARD );
      else 
        hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*) in_mat.DataDevice, (hipfftDoubleComplex*) out_mat.DataDevice, HIPFFT_BACKWARD );
      CUDA_LAST_ERROR();
    }else{
      mat = in_mat;
      if ( dire == MatrixOperations_FFT_FORWARD ){
        fftw_execute( cpu_plan_forward );
      }else{
        fftw_execute( cpu_plan_backward );
      }
      out_mat = mat;
    }
  }
  void execute(FloatComplexMatrix &in_mat,FloatComplexMatrix &out_mat,MatrixOperationsType dire)
  {
    if ( type == PlanType_float_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      if ( dire == MatrixOperations_FFT_FORWARD ) 
        hipfftExecC2C(gpu_plan, (hipfftComplex*) in_mat.DataDevice, (hipfftComplex*) out_mat.DataDevice, HIPFFT_FORWARD );
      else
        hipfftExecC2C(gpu_plan, (hipfftComplex*) in_mat.DataDevice, (hipfftComplex*) out_mat.DataDevice, HIPFFT_BACKWARD );
      CUDA_LAST_ERROR();
    }else{
      fmat = in_mat;
      if ( dire == MatrixOperations_FFT_FORWARD ){
        fftwf_execute( fcpu_plan_forward );
      }else{
        fftwf_execute( fcpu_plan_backward );
      }
      out_mat = fmat;
    }
  }
  void execute(Matrix &in_mat,ComplexMatrix &out_mat)
  {
    if ( type == PlanType_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      out_mat = in_mat;
      hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*) out_mat.DataDevice, (hipfftDoubleComplex*) out_mat.DataDevice, HIPFFT_FORWARD );
      CUDA_LAST_ERROR();
    }else{
      mat = in_mat;
      fftw_execute( cpu_plan_forward );
      out_mat = mat;
    }
  }
  void execute(FloatMatrix &in_mat,FloatComplexMatrix &out_mat)
  {
    if ( type == PlanType_float_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      out_mat = in_mat;
      hipfftExecC2C(gpu_plan, (hipfftComplex*) out_mat.DataDevice, (hipfftComplex*) out_mat.DataDevice, HIPFFT_FORWARD );
      CUDA_LAST_ERROR();
    }else{
      fmat = in_mat;
      fftwf_execute( fcpu_plan_forward );
      out_mat = fmat;
    }
  }
  void execute(ComplexMatrix &in_mat,Matrix &out_mat)
  {
    if ( type == PlanType_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      if ( not mat.SameDimensionQ( in_mat ) )
        mat.Init( in_mat );
      hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*) in_mat.DataDevice, (hipfftDoubleComplex*) mat.DataDevice, HIPFFT_BACKWARD);
      out_mat = mat;
      CUDA_LAST_ERROR();
    }else{
      mat = in_mat;
      fftw_execute( cpu_plan_backward );
      out_mat = mat;
    }
  }
  void execute(FloatComplexMatrix &in_mat,FloatMatrix &out_mat)
  {
    if ( type == PlanType_gpu_cpu and EvaSettings::MatrixPosition() == MatrixDevice ){
      if ( not fmat.SameDimensionQ( in_mat ) )
        fmat.Init( in_mat );
      hipfftExecC2C(gpu_plan, (hipfftComplex*) in_mat.DataDevice, (hipfftComplex*) fmat.DataDevice, HIPFFT_BACKWARD);
      out_mat = fmat;
      CUDA_LAST_ERROR();
    }else{
      fmat = in_mat;
      fftwf_execute( fcpu_plan_backward );
      out_mat = fmat;
    }
  }


};
static int planTablePtr;
static vector<planRecord*> planTable;


planRecord* getPlanReady(int rank, int n[],int type, int batch=1)
{
  if ( type == PlanType_mpi or type == PlanType_float_mpi )
    mpi_fft_init();
  bool found = false;
  planRecord *plan;
  //try find in table
  if ( planTable.size() > 0 ){
    if ( planTable[ planTablePtr ]->planQ(rank,n,type,batch) ){
      found = true;
    }else{
      planTablePtr = 0;
      while ( true  ){
        if ( planTable[ planTablePtr ]->planQ(rank,n,type,batch) ){
          found = true;
          break;
        }
        planTablePtr++;
        if ( planTablePtr  ==  (int)planTable.size()  )
          break;
      }
    }
  }
  // not find
  if ( not found ){
    plan = new planRecord(rank, n, type, batch );
    planTable.push_back( plan );
    return plan;
  }else
    return planTable[ planTablePtr ];
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, ComplexMatrix &in_mat, ComplexMatrix &out_mat, MatrixOperationsType direction)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat , direction );
  return;
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, FloatComplexMatrix &in_mat, FloatComplexMatrix &out_mat, MatrixOperationsType direction)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat , direction );
  return;
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, Matrix &in_mat, ComplexMatrix &out_mat)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat);
  return;
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, FloatMatrix &in_mat, FloatComplexMatrix &out_mat)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat);
  return;
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, ComplexMatrix &in_mat, Matrix &out_mat)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat);
  return;
}

void MatrixOperationsModule::FFT3D(int n1, int n2, int n3, FloatComplexMatrix &in_mat, FloatMatrix &out_mat)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  planRecord *plan;
  int n[3]={n1,n2,n3},batch;
  batch = in_mat.Size()/(n1*n2*n3);
  if ( batch*n1*n2*n3 != in_mat.Size() )
    ThrowError("FFT3D","Input matrix is not consistent with transformation size.");
  plan = getPlanReady(3,n,PlanType_gpu_cpu,batch);
  plan->execute( in_mat, out_mat);
  return;
}

void MatrixOperationsModule::MPI_FFT3D(int n1, int n2, int n3, ComplexMatrix &in_mat, ComplexMatrix &out_mat, MatrixOperationsType direction)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  if ( ( mpiModule->rankSize * in_mat.Size()) != (n1*n2*n3) )
    ThrowError("FFT3D","Matrix size does not match transformation size.");
  if ( ( mpiModule->rankSize * in_mat.D1 ) != n1 )
    ThrowError("FFT3D","Divided Matrix size of Dimension 1st does not match transformation size.");
  mpi_fft_init();
  planRecord *plan;
  int n[3]={n1,n2,n3};
  plan = getPlanReady(3,n,PlanType_mpi,1);
  if ( EvaSettings::MatrixPosition() == MatrixHost ){
    plan->execute( in_mat, out_mat , direction );
    out_mat.ReShape( 3, n1/mpiModule->rankSize, n2, n3 );
  }else{
    dprintf("acc plan apply in Device. Code not ready do nothing.\n");
  }
  return;
}

void MatrixOperationsModule::MPI_FFT3D(int n1, int n2, int n3, Matrix &in_mat, ComplexMatrix &out_mat)
{
  if ( not in_mat.SameDimensionQ( out_mat ) )
    ThrowError("FFT3D","Input matrix and output matrix should have same dimension.");
  if ( (mpiModule->rankSize * in_mat.Size()) != (n1*n2*n3) )
    ThrowError("FFT3D","Matrix size does not match transformation size.");
  if ( (mpiModule->rankSize * in_mat.D1 ) != n1 )
    ThrowError("FFT3D","Divided Matrix size of Dimension 1st does not match transformation size.");
  mpi_fft_init();
  planRecord *plan;
  int n[3]={n1,n2,n3};
  plan = getPlanReady(3,n,PlanType_mpi,1);
  plan->execute( in_mat, out_mat );
  out_mat.ReShape( 3, n1/mpiModule->rankSize, n2, n3 );
  return;
}

DefineFunction(fftTest)
{
  int nx,ny,nz;
  InitVariablePrint(nx,10);
  InitVariablePrint(ny,10);
  InitVariablePrint(nz,10);
  int n[3]={nx,ny,nz};
  FloatComplexMatrix in_data,out_data,data,data_hat;
  //debug data init
  if ( mpiModule->IsRootRank() ){
    in_data.Init(4,2,nx,ny,nz);
    for( int i=1;i<= nx; i++)
      for ( int j=1; j<=ny; j++)
        for ( int k=1; k<=nz; k++){
          in_data(0,i-1,j-1,k-1) = sin(i+j+k);
          in_data(1,i-1,j-1,k-1) = sin(i+j+k);
        }
    in_data.DumpFile("0.dat");
  } 

  ////////////////////////////////////
  //MatrixOperationsModule::MPI_FFT3D_MatrixInit( data, n[0],n[1],n[2] );
  // data = in_data;
  // data.HostToDevice();
  // EvaSettings::SetMatrixPosition(MatrixDevice);
  // FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_FORWARD );
  // data.DeviceToHost();
  // data.DumpFile("1.dat");
  // ////////////////////////////////////
  // FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_BACKWARD );
  // data.DeviceToHost();
  // data.DumpFile("2.dat");
  ReturnNull;
}



/*
DefineFunction(fftTest)
{
  Eva->mpiModule->Init();
  int nx,ny,nz;
  InitVariablePrint(nx,10);
  InitVariablePrint(ny,10);
  InitVariablePrint(nz,10);
  int n[3]={nx,ny,nz};
  ComplexMatrix in_data,out_data,data,data_hat;
  //debug data init
  if ( Eva->mpiModule->IsRootRank() ){
    in_data.Init(3,nx,ny,nz);
    for( int i=1;i<= nx; i++)
      for ( int j=1; j<=ny; j++)
        for ( int k=1; k<=nz; k++)
          in_data(i-1,j-1,k-1) = sin(i+j+k);
    in_data.DumpFile("in.dat");
  } 

  ////////////////////////////////////
  // same data with fftw
  fftw_mpi_init();
  data.Init(1,nx*ny*nz);
  fftw_plan plan = fftw_mpi_plan_dft_3d( nx, ny, nz, (fftw_complex*)data.Data, (fftw_complex*)data.Data, MPI_COMM_WORLD, FFTW_FORWARD, FFTW_ESTIMATE);  
  fftw_plan plan_back = fftw_mpi_plan_dft_3d( nx, ny, nz, (fftw_complex*)data.Data, (fftw_complex*)data.Data, MPI_COMM_WORLD, FFTW_BACKWARD, FFTW_ESTIMATE);  

  Eva->mpiModule->Distribute(in_data, data);

  fftw_execute( plan );

  //fftw_execute( plan_back );

  Eva->mpiModule->Collect( data,out_data );
  if ( Eva->mpiModule->IsRootRank() ){ 
    //cout<< out_data <<endl;
    out_data.DumpFile("2.dat");
  }

  ////////////////////////////////////
  //MatrixOperationsModule::MPI_FFT3D_MatrixInit( data, n[0],n[1],n[2] );
  Eva->mpiModule->Distribute(in_data, data, 1);
  data.HostToDevice();
  EvaSettings::SetMatrixPosition(MatrixDevice);
  FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_FORWARD );
  //MPI_FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_BACKWARD );
  EvaSettings::SetMatrixPosition(MatrixHost);
  data.DeviceToHost();
  Eva->mpiModule->Collect( data,out_data, 2);
  if ( Eva->mpiModule->IsRootRank() ){ 
    //cout<< out_data <<endl;
    out_data.DumpFile("1.dat");
  }



  ReturnNull;
}
*/

  
DefineFunction(MatrixMPI_FFT3DC2C){

  

  ReturnNull;
}


